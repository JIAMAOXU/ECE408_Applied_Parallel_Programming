#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 3
#define MASK_WIDTH 3
#define MASK_RADIUS 1
#define MASK_SIZE (TILE_WIDTH + (MASK_WIDTH - 1))
//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
 __shared__ float N_ds[MASK_SIZE][MASK_SIZE][MASK_SIZE];

  // Shifting from output coordinates to input coordinates 
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int bz = blockIdx.z;
  int x_col_o = bx * TILE_WIDTH + tx;
  int y_row_o = by * TILE_WIDTH + ty;
  int z_height_o = bz * TILE_WIDTH + tz;

  int x_col_i = x_col_o - MASK_RADIUS;
  int y_row_i = y_row_o - MASK_RADIUS;
  int z_height_i = z_height_o - MASK_RADIUS;

// __shared__ float N_ds[TILE_SIZE+MAX_MASK_WIDTH-1][TILE_SIZE+MAX_MASK_HEIGHT-1];
// If ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width)) {
//  N_ds[ty][tx] = data[row_i * pitch + col_i];
// } else{
//  N_ds[ty][tx] = 0.0f;
// }

// Taking Care of Boundaries
if (
      (x_col_i >= 0)    &&  (x_col_i < x_size)    &&
      (y_row_i >= 0)    &&  (y_row_i < y_size)    &&
      (z_height_i >= 0) &&  (z_height_i < z_size)
    )
    {N_ds[tz][ty][tx] = input[z_height_i * y_size * x_size + y_row_i * x_size + x_col_i];}
else
    {N_ds[tz][ty][tx] = 0.0f;}

__syncthreads(); // wait for tile

// Not All Threads Calculate Output
  float Pvalue = 0.0f;
  if(tz < TILE_WIDTH && ty < TILE_WIDTH && tx < TILE_WIDTH ){
    for(int i = 0; i < MASK_WIDTH; i++) { 
      for(int j = 0; j < MASK_WIDTH; j++) {
        for (int k = 0; k < MASK_WIDTH; k++) {
          Pvalue += Mc[i][j][k] * N_ds[i+tz][j+ty][k+tx];  
        }
      }
    }
    if(x_col_o < x_size && y_row_o < y_size &&  z_height_o < z_size) {
      output[z_height_o * y_size * x_size + y_row_o * x_size + x_col_o] = Pvalue; 
    }
  }
}


int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void**) &deviceInput,(inputLength - 3)*sizeof(float));
  hipMalloc((void**) &deviceOutput,(inputLength - 3)*sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");


  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mc), hostKernel, kernelLength * sizeof(float));
  wbTime_stop(Copy, "Copying data to the GPU");


  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(((float)x_size) / TILE_WIDTH), 
               ceil(((float)y_size) / TILE_WIDTH), 
               ceil(((float)z_size) / TILE_WIDTH));
  dim3 DimBlock(MASK_SIZE, MASK_SIZE, MASK_SIZE);
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
   hipMemcpy(hostOutput+3, deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
